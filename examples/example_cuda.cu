#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include <vector>

#include <hip/hip_runtime.h>

#include <tracer/cuda>

DEFINE_TRACE_CONTEXTS;

template <typename R>
__global__ void kern(R *x, int n)
{
    for (int i = 0; i < n; ++i) { x[i] = i; }
}

template <typename R = int>
void f(int n)
{
    TRACE_SCOPE(__func__);
    void *data;
    hipMalloc(&data, n * sizeof(R));
    R *x = (R *)data;
    kern<R><<<1, 1>>>(x, n);
    std::vector<R> y(n);
    hipMemcpy(y.data(), data, n * sizeof(R), hipMemcpyDeviceToHost);
    hipFree(data);

    const R sum = std::accumulate(y.begin(), y.end(), 0);
    std::cout << sum << std::endl;
}

int main()
{
    TRACE_SCOPE(__func__);
    for (int i = 0; i < 10; ++i) { f((i + 1) * 1024); }
    return 0;
}
